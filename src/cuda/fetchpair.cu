
#include <hip/hip_runtime.h>

// #include "linalg.cu"






/*!
 * Compute the initial labels for a gene pair in an expression matrix. Samples
 * with missing values and samples that fall below the expression threshold are
 * labeled as such, all other samples are labeled as cluster 0. The number of
 * clean samples is returned.
 *
 * @param globalWorkSize
 * @param expressions
 * @param sampleSize
 * @param in_index
 * @param minExpression
 * @param out_N
 * @param out_labels
 */
__global__
void fetchPair(
   int globalWorkSize,
   const float *expressions,
   int sampleSize,
   const int2 *in_index,
   int minExpression,
   int *out_N,
   char *out_labels)
{
   int i = blockIdx.x * blockDim.x + threadIdx.x;

   if ( i >= globalWorkSize )
   {
      return;
   }

   // initialize variables
   int2 index = in_index[i];
   char *labels = &out_labels[i * sampleSize];
   int *p_N = &out_N[i];

   // index into gene expressions
   const float *gene1 = &expressions[index.x * sampleSize];
   const float *gene2 = &expressions[index.y * sampleSize];

   // populate X with shared expressions of gene pair
   int N = 0;

   for ( int i = 0; i < sampleSize; ++i )
   {
      if ( isnan(gene1[i]) || isnan(gene2[i]) )
      {
         labels[i] = -9;
      }
      else if ( gene1[i] < minExpression || gene2[i] < minExpression )
      {
         labels[i] = -6;
      }
      else
      {
         N++;
         labels[i] = 0;
      }
   }

   // save number of clean samples
   *p_N = N;
}