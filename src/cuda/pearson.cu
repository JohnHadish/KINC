
#include <hip/hip_runtime.h>






/*!
 * Compute the Pearson correlation of a cluster in a pairwise data array.
 *
 * @param x
 * @param y
 * @param labels
 * @param sampleSize
 * @param cluster
 * @param minSamples
 */
__device__
float Pearson_computeCluster(
   const float *x,
   const float *y,
   const char *labels,
   int sampleSize,
   char cluster,
   int minSamples)
{
   // compute intermediate sums
   int n = 0;
   float sumx = 0;
   float sumy = 0;
   float sumx2 = 0;
   float sumy2 = 0;
   float sumxy = 0;

   for ( int i = 0; i < sampleSize; ++i )
   {
      if ( labels[i] == cluster )
      {
         float x_i = x[i];
         float y_i = y[i];

         sumx += x_i;
         sumy += y_i;
         sumx2 += x_i * x_i;
         sumy2 += y_i * y_i;
         sumxy += x_i * y_i;

         ++n;
      }
   }

   // compute correlation only if there are enough samples
   float result = NAN;

   if ( n >= minSamples )
   {
      result = (n*sumxy - sumx*sumy) / sqrt((n*sumx2 - sumx*sumx) * (n*sumy2 - sumy*sumy));
   }

   return result;
}






/*!
 * Compute the correlation of each cluster in a pairwise data array. The data array
 * should only contain the clean samples that were extracted from the expression
 * matrix, while the labels should contain all samples.
 *
 * @param globalWorkSize
 * @param expressions
 * @param sampleSize
 * @param in_index
 * @param clusterSize
 * @param in_labels
 * @param minSamples
 * @param out_correlations
 */
__global__
void Pearson_compute(
   int globalWorkSize,
   const float *expressions,
   int sampleSize,
   const int2 *in_index,
   char clusterSize,
   const char *in_labels,
   int minSamples,
   float *out_correlations)
{
   int i = blockIdx.x * blockDim.x + threadIdx.x;

   if ( i >= globalWorkSize )
   {
      return;
   }

   // initialize workspace variables
   int2 index = in_index[i];
   const float *x = &expressions[index.x * sampleSize];
   const float *y = &expressions[index.y * sampleSize];
   const char *labels = &in_labels[i * sampleSize];
   float *correlations = &out_correlations[i * clusterSize];

   for ( char k = 0; k < clusterSize; ++k )
   {
      correlations[k] = Pearson_computeCluster(x, y, labels, sampleSize, k, minSamples);
   }
}
